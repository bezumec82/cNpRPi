#include <sys/utsname.h>

// Includes, system
#include <stdio.h>
#include <cassert>

// Includes CUDA
#include <hip/hip_runtime.h>

const char *sampleName = "simpleAssert";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Tests assert function.
//! Thread whose id > N will print assertion failed error message.
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(int N)
{
    int gtid = blockIdx.x*blockDim.x + threadIdx.x ;
    assert(gtid < N) ;
}

void simpleAssert( void )
{
    int Nblocks = 2;
    int Nthreads = 32;
    hipError_t error ;

    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    dim3 dimGrid(Nblocks);
    dim3 dimBlock(Nthreads);

    printf("Launch kernel to generate assertion failures\n");
    testKernel<<<dimGrid, dimBlock>>>(60);

    //Synchronize (flushes assert output).
    printf("\n-- Begin assert output\n\n");
    error = hipDeviceSynchronize();
    printf("\n-- End assert output\n\n");

    //Check for errors and failed asserts in asynchronous kernel launch.
    if (error == hipErrorAssert)
    {
        printf("Device assert failed as expected, "
               "CUDA error message is: %s\n\n",
               hipGetErrorString(error));
    }


    testResult = error == hipErrorAssert;
}