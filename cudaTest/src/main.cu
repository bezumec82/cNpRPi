#include "hip/hip_runtime.h"

#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <iterator>

using namespace std;

__global__ void add_block(int * a, int * b, int * sum)
{
    //Use 'blockIdx.x' to access block index
    sum[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
/*--- End of the function ----------------------------------------------------*/

__global__ void add_thread(int * a, int * b, int * sum)
{
    //Use 'blockIdx.x' to access block index
    sum[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}
/*--- End of the function ----------------------------------------------------*/

__global__ void add_mixed(int * a, int * b, int * sum)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    sum[index] = a[index] + b[index];
}
/*--- End of the function ----------------------------------------------------*/

__global__ void add_safe(int *a, int * b, int * sum, int arr_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < arr_size)
    {
        sum[index] = a[index] + b[index];
    }
}
/*--- End of the function ----------------------------------------------------*/

void random_ints(int *arr_ptr, int arr_size)
{
   for (int i = 0; (i < arr_size)&&arr_ptr; ++i, arr_ptr++)
       *arr_ptr = rand() % 100;
}
/*--- End of the function ----------------------------------------------------*/

#define THREADS                 2
#define BLOCKS                  2
#define SIZE                    THREADS * BLOCKS

#define BLOCK_IMPLEMENTATION    false
#define MIXED_IMPLEMENTATION    false
#define SAFE_IMPLEMENTATION     false
int main(void) {

    //Host copies
	int     a   [SIZE] = {0};
	int     b   [SIZE] = {0};
	int     sum [SIZE] = {0};

    //Setup values
	random_ints(a, SIZE);
	random_ints(b, SIZE);

    for (auto iter : a     ){ cout << iter << " "; };  cout <<endl;
    for (auto iter : b     ){ cout << iter << " "; };  cout <<endl;
	//Device copies
	int * devp_a, * devp_b, * devp_sum;

	//Allocate space for device copies
	hipMalloc((void **)&devp_a,    SIZE * sizeof(int));
    hipMalloc((void **)&devp_b,    SIZE * sizeof(int));
    hipMalloc((void **)&devp_sum,  SIZE * sizeof(int));

	//Copy inputs to the device
	hipMemcpy(devp_a,  a,          SIZE * sizeof(int),   hipMemcpyHostToDevice);
    hipMemcpy(devp_b,  b,          SIZE * sizeof(int),   hipMemcpyHostToDevice);

    //Launch N copies of 'add()' with 'add<<<N,1>>>(...)'
#if BLOCK_IMPLEMENTATION
    add_block<<<SIZE,1>>>(devp_a, devp_b, devp_sum);
#elif THREAD_IMPLEMENTATION
    add_thread<<<1,SIZE>>>(devp_a, devp_b, devp_sum);
#elif MIXED_IMPLEMENTATION
    add_mixed<<<BLOCKS,THREADS>>>(devp_a, devp_b, devp_sum);
#elif SAFE_IMPLEMENTATION
    add_safe<<<BLOCKS,THREADS>>>(devp_a, devp_b, devp_sum, SIZE * sizeof(int));
#endif

    //Copy result back to the host
    hipMemcpy(&sum,    devp_sum,   SIZE * sizeof(int),   hipMemcpyDeviceToHost );



    cout << "Result : " << endl;
    std::copy   (
                std::begin(sum),
                std::end(sum),
                std::ostream_iterator<int>(std::cout, " ")
                );

    //Cleanup
    hipFree(devp_a);
    hipFree(devp_b);
    hipFree(devp_sum);

	return 0;
}
